#include "hip/hip_runtime.h"
#include "tricount.h"
#include "hip/hip_runtime.h"
#include <iostream>

using namespace std;

const int WARPSIZE = 16;
const int BLOCKSIZE = 32;
uint64_t edgeBlockSize = 1024 * 1024 * 1024 / sizeof(uint64_t) * 8;


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


void initGPU(const uint64_t edge_num, const uint32_t N){
    hipDeviceProp_t deviceProp;
    gpuErrchk( hipGetDeviceProperties(&deviceProp, 0) );
    
    // 64MB for redundancy
    uint64_t usageMem = edge_num / 2 * 8 + (N + 1) * 4 * 3 + 1024 * 1024 * 64;
    if(usageMem > deviceProp.totalGlobalMem){
        cerr << "Global memory(" 
                << deviceProp.totalGlobalMem / 1024 / 1024 
                << "MB) is not enough. Require " 
                << usageMem / 1024 / 1024 << "MB" << endl;
        exit(2);
    }

    edgeBlockSize = (deviceProp.totalGlobalMem - usageMem) / sizeof(Edge_t);
    edgeBlockSize = min(edge_num, edgeBlockSize);

#ifdef DEBUG_DF
    cout << "Global memory: " 
                << deviceProp.totalGlobalMem / 1024 / 1024 
                << "MB; Require: " 
                << (usageMem + edgeBlockSize * sizeof(Edge_t)) / 1024 / 1024 << "MB." << endl;
#endif
}


__device__ void intersection16(const uint32_t* lbases, const uint32_t* rbases, uint32_t ln, uint32_t rn, 
                                unsigned long long* p_mysum){
    __shared__ uint32_t lblock[BLOCKSIZE];
    __shared__ uint32_t rblock[BLOCKSIZE];

    const int warpBegin = threadIdx.x & (~(WARPSIZE - 1));
    const int threadLane = threadIdx.x & (WARPSIZE - 1);

    uint32_t i = 0, j = 0, sum = 0;
    uint32_t lsize = WARPSIZE, rsize = WARPSIZE;

    while (i < ln && j < rn) {

        lsize = min(ln - i, WARPSIZE);
        rsize = min(rn - j, WARPSIZE);

        if(i + threadLane < ln) lblock[threadIdx.x] = lbases[i + threadLane];
        if(j + threadLane < rn) rblock[threadIdx.x] = rbases[j + threadLane];

        __threadfence_block();

        for(int k = 0; k < rsize; ++k)
            sum += (threadLane < lsize) & (lblock[threadIdx.x] == rblock[warpBegin + k]);

        uint32_t llast = lblock[warpBegin + lsize - 1];
        uint32_t rlast = rblock[warpBegin + rsize - 1];

        if(llast >= rlast) j += rsize;
        if(llast <= rlast) i += lsize;
    }

    (*p_mysum) += sum;
}


__device__ unsigned long long dev_sum;
__device__ unsigned int dev_nowNode;


__global__ void __tricount(uint32_t N, const uint32_t* __restrict__ nodeIndex, const uint32_t* __restrict__ adjList){
    __shared__ unsigned long long sdata[BLOCKSIZE];
    unsigned long long mysum = 0;

    const int warpLane = threadIdx.x / WARPSIZE;
    const int warpNum = blockDim.x / WARPSIZE;

    __shared__ unsigned int nodeI;
    __shared__ unsigned int nodeEnd;
    

    while(true){
        if(threadIdx.x == 0){
            if(++nodeI >= nodeEnd){
                nodeI = atomicAdd(&dev_nowNode, 256);
                nodeEnd = min(N, nodeI + 256);
            }
        }

        __syncthreads();

        unsigned int i = nodeI;
        if(i >= N) break;

        uint32_t lb = nodeIndex[i];
        uint32_t le = nodeIndex[i + 1];
        uint32_t ln = le - lb;

        for(uint32_t j = lb + warpLane; j < le; j += warpNum){
            uint32_t ri = adjList[j];
            uint32_t rn = nodeIndex[ri+1] - nodeIndex[ri];
            uint32_t rb = nodeIndex[ri];

            intersection16(adjList + lb, adjList + rb, ln, rn, &mysum);
        }
    }

    sdata[threadIdx.x] = mysum;
    __syncthreads();

    for (int s=1; s < blockDim.x; s *=2){
        int index = 2 * s * threadIdx.x;

        if (index < blockDim.x){
            sdata[index] += sdata[index + s];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0)
        atomicAdd(&dev_sum, sdata[0]);
}


unsigned long long tricount(uint32_t N, const uint32_t* nodeIndex, const uint32_t* adjList){
    int numBlocks = 2048;

    uint32_t* dev_nodeIndex;
    uint32_t* dev_adjList;

    uint64_t size_nodeIndex = sizeof(uint32_t) * (N + 1);
    uint64_t size_adjList = sizeof(uint32_t) * nodeIndex[N];

    gpuErrchk( hipMalloc((void**)&dev_nodeIndex, size_nodeIndex) );
    gpuErrchk( hipMalloc((void**)&dev_adjList, size_adjList) );

    // copy inputs to device
    gpuErrchk( hipMemcpy(dev_nodeIndex, nodeIndex, size_nodeIndex, hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(dev_adjList, adjList, size_adjList, hipMemcpyHostToDevice) );


    unsigned long long sum = 0;
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(dev_sum), &sum, sizeof(unsigned long long)) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(dev_nowNode), &sum, sizeof(unsigned int)) );

    __tricount<<<numBlocks, BLOCKSIZE>>>(N, dev_nodeIndex, dev_adjList);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    gpuErrchk( hipMemcpyFromSymbol(&sum, HIP_SYMBOL(dev_sum), sizeof(sum)) );

    return sum;
}


// degree collect on gpu
__global__ void __alldegreeCollect(const Edge_t* __restrict__ edgeList, const uint64_t __restrict__ edge_num, 
                                    uint32_t* deg){

    int blockSize = blockDim.x * gridDim.x;
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for(uint64_t i=tid; i<edge_num; i += blockSize){
        const Edge_t &e = edgeList[i];

        int res = (e.u != e.v);
        atomicAdd(deg + e.u, res);
        atomicAdd(deg + e.v, res);
    }
}

__global__ void __outdegreeCollect(const Edge_t* __restrict__ edgeList, const uint64_t __restrict__ edge_num, 
                                    uint32_t* deg){

    int blockSize = blockDim.x * gridDim.x;
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for(uint64_t i=tid; i<edge_num; i += blockSize){
        const Edge_t &e = edgeList[i];

        atomicAdd(deg + e.u, e.u != e.v);
    }
}

void degreeCollect(const int type, const Edge_t* edgeList, const uint64_t edge_num, 
                    uint32_t* deg, uint32_t N){
    int numBlocks = 2048;

    Edge_t* dev_edgeList;
    uint32_t* dev_deg;

    gpuErrchk( hipMalloc((void**)&dev_edgeList, edgeBlockSize * sizeof(Edge_t)) );
    gpuErrchk( hipMalloc((void**)&dev_deg, N * sizeof(uint32_t)) );

    auto func = __alldegreeCollect;
    if(type == 1){
        func = __outdegreeCollect;
    }

    for(uint64_t i = 0; i < edge_num; i += edgeBlockSize){
        uint64_t copySize = min(edge_num - i, edgeBlockSize);

        gpuErrchk( hipMemcpy(dev_edgeList, edgeList + i, copySize * sizeof(Edge_t), hipMemcpyHostToDevice) );
        func<<<numBlocks, BLOCKSIZE>>>(dev_edgeList, copySize, dev_deg);
        gpuErrchk( hipDeviceSynchronize() );
    }

    gpuErrchk( hipMemcpy(deg, dev_deg, N * sizeof(uint32_t), hipMemcpyDeviceToHost) );

    hipFree(dev_edgeList);
    hipFree(dev_deg);
}


// adjList(CSR) construct on gpu
__global__ void __adjListConstruct(const Edge_t* edgeList, const uint64_t edge_num, 
                        const uint32_t* nbr_u, uint32_t* nbr_size, uint32_t* nbr_arr){

    int blockSize = blockDim.x * gridDim.x;
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for(uint64_t i=tid; i<edge_num; i += blockSize){
        const uint32_t &u = edgeList[i].u;
        const uint32_t &v = edgeList[i].v;
        
        if(u == v) continue;

        uint32_t j = atomicAdd(nbr_size + u, 1);
        nbr_arr[nbr_u[u] + j] = v;
    }
}

void adjListConstruct(const Edge_t* edgeList, const uint64_t edge_num, 
                        const uint32_t* nbr_u, uint32_t* nbr_arr, const uint32_t N){
    int numBlocks = 2048;

    Edge_t* dev_edgeList;
    uint32_t* dev_nbr_arr;
    uint32_t* dev_nbr_u;
    uint32_t* dev_nbr_size;

    gpuErrchk( hipMalloc((void**)&dev_edgeList, edgeBlockSize * sizeof(Edge_t)) );
    gpuErrchk( hipMalloc((void**)&dev_nbr_arr, edge_num * sizeof(uint32_t)) );
    gpuErrchk( hipMalloc((void**)&dev_nbr_u, N * sizeof(uint32_t)) );
    gpuErrchk( hipMalloc((void**)&dev_nbr_size, N * sizeof(uint32_t)) );

    gpuErrchk( hipMemcpy(dev_nbr_u, nbr_u, N * sizeof(uint32_t), hipMemcpyHostToDevice) );

    for(uint64_t i = 0; i < edge_num; i += edgeBlockSize){
        uint64_t copySize = min(edge_num - i, edgeBlockSize);

        gpuErrchk( hipMemcpy(dev_edgeList, edgeList + i, copySize * sizeof(Edge_t), hipMemcpyHostToDevice) );
        __adjListConstruct<<<numBlocks, BLOCKSIZE>>>(dev_edgeList, copySize, 
                                                        dev_nbr_u, dev_nbr_size, dev_nbr_arr);
        gpuErrchk( hipDeviceSynchronize() );
    }

    gpuErrchk( hipMemcpy(nbr_arr, dev_nbr_arr, edge_num * sizeof(uint32_t), hipMemcpyDeviceToHost) );

    hipFree(dev_edgeList);
    hipFree(dev_nbr_arr);
    hipFree(dev_nbr_u);
    hipFree(dev_nbr_size);
}

